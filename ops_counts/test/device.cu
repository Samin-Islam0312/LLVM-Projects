
#include <hip/hip_runtime.h>
// device_module.cu (or your device TU)
extern "C" {
__device__ unsigned long long gBinOpCounts[6];  // 0..5

// 0=FAdd, 1=FSub, 2=FMul, 3=FDiv, 4=FRem, 5=FMA (if you enable it)
__device__ void __record_binop(int id) {
  if (id < 0 || id > 5) return;
  atomicAdd(&gBinOpCounts[id], 1ULL);
}

// your kernel stays the same
extern "C" __global__
void divKernel(const float* A, const float* B, float* C, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    float x = A[i], y = B[i];

    // 1 × FMA
    float r = fmaf(x, y, 1.0f);

    // 3 × add
    r = r + 1.0f;
    r = r + 2.0f;
    r = r + 3.0f;

    // 8 × sub
    r = r - 1.0f; r = r - 2.0f; r = r - 3.0f; r = r - 4.0f;
    r = r - 5.0f; r = r - 6.0f; r = r - 7.0f; r = r - 8.0f;

    // keep your division (1 × fdiv)
    C[i] = r / (y + 1e-20f);
  }
}

} // extern "C"
