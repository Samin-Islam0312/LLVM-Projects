// file: host_driver.cpp
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <fstream>
#include <iostream>
#include <string>

static void ck(hipError_t r, const char* where) {
  if (r != hipSuccess) {
    const char *name=nullptr, *str=nullptr;
    hipDrvGetErrorName(r, &name);
    hipDrvGetErrorString(r, &str);
    std::fprintf(stderr, "CUDA Driver error at %s: %s (%s)\n",
                 where, name?name:"?", str?str:"?");
    std::exit(1);
  }
}

static std::string load_text_file(const char* path) {
  std::ifstream ifs(path, std::ios::in | std::ios::binary);
  if (!ifs) { std::perror(path); std::exit(1); }
  return std::string((std::istreambuf_iterator<char>(ifs)),
                     std::istreambuf_iterator<char>());
}

int main(int argc, char** argv) {
  if (argc < 2) {
    std::fprintf(stderr, "Usage: %s <instrumented.ptx> [N=1048576] [block=256]\n", argv[0]);
    return 1;
  }
  const char* ptxPath = argv[1];
  int N = (argc >= 3) ? std::atoi(argv[2]) : (1<<20);
  int block = (argc >= 4) ? std::atoi(argv[3]) : 256;
  if (N <= 0) N = 1<<20;
  if (block <= 0) block = 256;
  int grid = (N + block - 1) / block;

  ck(hipInit(0), "hipInit");
  hipDevice_t dev; ck(hipDeviceGet(&dev, 0), "hipDeviceGet");
  hipCtx_t ctx; ck(hipCtxCreate(&ctx, 0, dev), "hipCtxCreate");

  // Load instrumented PTX
  std::string ptx = load_text_file(ptxPath);
  hipModule_t mod; ck(hipModuleLoadDataEx(&mod, ptx.c_str(), 0, nullptr, nullptr), "hipModuleLoadDataEx");

  // Resolve kernel and globals (names must match your device file)
  hipFunction_t k; ck(hipModuleGetFunction(&k, mod, "divKernel"), "hipModuleGetFunction(divKernel)");

  // Per-op counters: gBinOpCounts[6] (u64)
  hipDeviceptr_t dCounts; size_t countsBytes = 0;
  ck(hipModuleGetGlobal(&dCounts, &countsBytes, mod, "gBinOpCounts"), "hipModuleGetGlobal(gBinOpCounts)");
  if (countsBytes < 6 * sizeof(unsigned long long)) {
    std::fprintf(stderr, "gBinOpCounts size mismatch: %zu\n", countsBytes);
    return 1;
  }
  ck(hipMemsetD8(dCounts, 0, 6*sizeof(unsigned long long)), "reset gBinOpCounts");

  // Buffers
  size_t bytes = static_cast<size_t>(N) * sizeof(float);
  hipDeviceptr_t dA, dB, dC;
  ck(hipMalloc(&dA, bytes), "hipMalloc(A)");
  ck(hipMalloc(&dB, bytes), "hipMalloc(B)");
  ck(hipMalloc(&dC, bytes), "hipMalloc(C)");

  std::vector<float> hA(N), hB(N);
  for (int i=0;i<N;i++){ hA[i]=float(i+1); hB[i]=float((i%7)+1); }
  ck(hipMemcpyHtoD(dA, hA.data(), bytes), "hipMemcpyHtoD(A)");
  ck(hipMemcpyHtoD(dB, hB.data(), bytes), "hipMemcpyHtoD(B)");

  // Launch
  void* params[] = { &dA, &dB, &dC, &N };
  ck(hipModuleLaunchKernel(k,
                    grid, 1, 1,
                    block, 1, 1,
                    0, 0, params, nullptr),
     "hipModuleLaunchKernel");
  ck(hipCtxSynchronize(), "hipCtxSynchronize");

  // Read back per-op counts
  unsigned long long hCounts[6] = {};
  ck(hipMemcpyDtoH(hCounts, dCounts, 6*sizeof(unsigned long long)), "hipMemcpyDtoH(gBinOpCounts)");

  const char* names[6] = {"FAdd","FSub","FMul","FDiv","FRem","FMA"};
  std::puts("Per-opcode FP binary counts:");
  for (int i=0;i<6;i++)
    std::printf("  %-4s = %llu\n", names[i], (unsigned long long)hCounts[i]);

  // (Optional) pull result to verify math path executed
  std::vector<float> hC; hC.resize(N);
  ck(hipMemcpyDtoH(hC.data(), dC, bytes), "hipMemcpyDtoH(C)");

  hipFree(dA); hipFree(dB); hipFree(dC);
  hipModuleUnload(mod);
  hipCtxDestroy(ctx);
  return 0;
}
